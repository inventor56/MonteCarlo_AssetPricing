#include "hip/hip_runtime.h"
#include "kernel_cuda.h"
#include "book.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "MonteCarlo.h"

__global__ void kernel(double drift, double init_price, int days, float* normals, double* result) {
  int index = blockIdx.x*gridDim.x;
  double currentPrice = init_price; // Get last value in price data to start with.
  for (int i = 0; i < days; i++) {
    /*
    result[index+i] = currentPrice*(exp(drift+normals[index+i]));
    currentPrice = result[index+i];
    */
    result[index+i] = currentPrice+1;
    currentPrice = result[index+i];
    printf("Ok, our normals are at: %f \n", normals[index+i]);
    //printf("Ok at Index %d, we have %f \n", index+i, currentPrice);
  }
  //printf("The block id is %d \n", blockIdx.x);
  //printf("Ok at Index %d", index);
  //printf("the grid dimesnions are %d \n", gridDim.x);
  //result[3] = init_price*(exp(drift+normals[4]));
}

double** cuda_run(double* histArr, int histLength, int daysToGenerate, int simulationsToRun) {
  int n_size = daysToGenerate*simulationsToRun;
  double drift_amt;

  // Allocate memory for...
  // Random Numbers
  float* rand_normals; //= new float[n_size]


  // Array (one-dimensional) in which we will store our reuslts in intially, before converting
  double* initial_cuda_results;
  HANDLE_ERROR( hipMallocManaged(&rand_normals, n_size * sizeof(float)));
  HANDLE_ERROR( hipMallocManaged(&initial_cuda_results, n_size * sizeof(double)));

  // Array in which we will store our final results in
  double ** cuda_results = new double*[simulationsToRun];
  for (int r = 0; r < simulationsToRun; r++) {
    cuda_results[r] = new double[daysToGenerate];
  }


  // Calculate drift
  MonteCarlo* createDrift = new MonteCarlo(histArr, histLength, daysToGenerate, simulationsToRun);
  createDrift->calculateResults(false); // cuda version, just calculateto get the drift
  drift_amt = createDrift->getDrift();

  // Create all random numbers for the normal distribution, using currand parallelism
  hiprandGenerator_t curGen;
  hiprandCreateGenerator(&curGen, HIPRAND_RNG_PSEUDO_MTGP32); // use the mersenne twister algorithm for HIPRAND_RNG_PSEUDO_MTGP32
  hiprandSetPseudoRandomGeneratorSeed(curGen, 1234ULL); // generate using a big unsigned int as your seed
  hiprandGenerateNormal(curGen, rand_normals, n_size, 0.0f, 1.0f); // generate
  hiprandDestroyGenerator(curGen);

  // Now we can run our parallel program
  kernel<<<simulationsToRun,1>>>(drift_amt, histArr[histLength-1], daysToGenerate, rand_normals, initial_cuda_results);

  // Barrier to wait for computations to complete on the GPU before proceeding
  hipDeviceSynchronize();

  // Convert results to 2d array
  for (int g = 0; g < simulationsToRun; g++) {
    for (int k = 0; k < daysToGenerate; k++) {
      cuda_results[g][k] = initial_cuda_results[(k*simulationsToRun)+g];
    }
  }


  // Free up memory on the GPU
  HANDLE_ERROR (hipFree(rand_normals));
  HANDLE_ERROR (hipFree(initial_cuda_results));

  //Return results
  return cuda_results;
}
